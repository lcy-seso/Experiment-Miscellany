#include "hip/hip_runtime.h"
#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "kernels/generated_kernels.h"
#include "kernels/kernel_func_def.h"
#include "utils.h"

#define cudaErrCheck(stat) \
  { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char* file, int line) {
  if (stat != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file,
            line);
  }
}

void fillRandom(float* A, int elementNum) {
  // create a pseudo-random number generator
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

  // set the seed for the random number generator using the system clock
  hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

  // fill the array with random numbers on the device
  hiprandGenerateUniform(prng, A, elementNum);
}

void checkResults(const float* C1, const float* C2, int numel) {
  float* C1_cpu = (float*)malloc(numel * sizeof(float));
  float* C2_cpu = (float*)malloc(numel * sizeof(float));
  hipMemcpy(C1_cpu, C1, numel * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(C2_cpu, C2, numel * sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < numel; ++i) {
    std::cout << C1_cpu[i] << std::endl;
  }

  free(C1_cpu);
  free(C2_cpu);
}

void createData(int m, int n, int k, float* d_A, float* d_B, float* d_C1,
                float* d_C2) {
  int64_t size_A = m * k;
  int64_t size_B = k * n;
  int64_t size_C = m * n;

  hipMalloc(&d_A, size_A * sizeof(float));
  hipMalloc(&d_B, size_B * sizeof(float));

  hipMalloc(&d_C1, size_C * sizeof(float));
  hipMalloc(&d_C2, size_C * sizeof(float));

  fillRandom(d_A, size_A);
  fillRandom(d_B, size_B);
  fillZeros(d_C1, size_C);
  fillZeros(d_C2, size_C);
}

void testAutoGenGemm(int m, int n, int k, dim3 grid, dim3 block,
                     KernelFunc func, std::vector<float>& info) {
  float* d_A = NULL;
  float* d_B = NULL;
  float* d_C1 = NULL;
  float* d_C2 = NULL;
  // allocate GPU memory
  createData(m, n, k, d_A, d_B, d_C1, d_C2);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float total1 = 0.;
  float elapsed = 0.;
  int iters = 20;
  for (int i = 0; i < iters; ++i) {
    hipEventRecord(start, 0);

    func<<<grid, block>>>(d_A, d_B, d_C1);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);

    total1 += elapsed;
  }

  // CuBlas
  float total2 = 0.;
  float alf = 1.0, bet = 0.0;
  const float* alpha = &alf;
  const float* beta = &bet;
  elapsed = 0.;

  hipblasHandle_t handle;
  // create cuBlas handler
  hipblasCreate(&handle);
  hipblasSgemm(handle, HIPBLAS_OP_N /* transA */, HIPBLAS_OP_N /* transB */, m, n,
              k, alpha, d_A, m, d_B, k, beta, d_C2, m);

  for (int i = 0; i < iters; ++i) {
    hipEventRecord(start, 0);

    hipblasSgemm(handle, HIPBLAS_OP_N /* transA */, HIPBLAS_OP_N /* transB */, m,
                n, k, alpha, d_A, m, d_B, k, beta, d_C2, m);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsed, start, stop);
    total2 += elapsed;
  }

  checkResults(d_C1, d_C2, m * n);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C1);
  hipFree(d_C2);

  info[0] = total1 / iters;
  info[1] = total2 / iters;
}

int main(int argc, char** argv) {
  createKernelInfoMap(kernelMap);

  std::cout.setf(std::ios::fixed);
  std::cout << std::setprecision(4);
  std::cout << "|[m, n, k]|Generated Kernels|cuBlas|" << std::endl
            << "|:--|:--|:--|" << std::endl;

  auto it = kernelMap.begin();
  while (it != kernelMap.end()) {
    std::vector<float> times(2, 0.);
    auto func = it->second;
    testAutoGenGemm(func.m, func.n, func.k, *(func.grid), *(func.block),
                    func.func, times);
    it++;

    std::cout << "[" << func.m << ", " << func.n << ", " << func.k << "]|"
              << times[0] << "|" << times[1] << "|" << std::endl;
  }

  return 0;
}
